#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <iostream>

//Thrust includes
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/iterator/counting_iterator.h>

#include "cuda_helpers.cuh"

typedef uint8_t U8;
typedef uint32_t U32;

using namespace thrust;

void mark_head(thrust::device_vector<int>& keys, thrust::device_vector<int>& buckets){
	
	int *keys_r = raw(keys);
	int *bucket_r = raw(buckets);
	auto r = counting_iterator<int>(0);
	int n = keys.size();

	for_each(r, r + n, [=] __device__(int i) {
		//First item is always head
		if (i == 0){
			bucket_r[i] = 1;
		}
		//Is different than previous item - must be a bucket head
		else if (keys_r[i] != keys_r[i - 1]){
			bucket_r[i] = 1;
		}
		//Not a bucket head
		else{
			bucket_r[i] = 0;
		}
	});
}


void get_rank(thrust::device_vector<int>& buckets, thrust::device_vector<int>& rank, thrust::device_vector<int>& sa){

	//Scan bucket heads into keys. Keys just used as temporary storage
	inclusive_scan(buckets.begin(), buckets.end(), buckets.begin());
	
	//Calculate rank - stores rank inverse to the suffix array
	// e.g. rank[3] stores the bucket position of sa[?] = 3
	int *rank_r = raw(rank);
	int *sa_r = raw(sa);
	int *buckets_r = raw(buckets);

	auto r = counting_iterator<int>(0);
	int n = sa.size();

	for_each(r, r + n, [=] __device__(int i) {
		int suffix = sa_r[i];
		rank_r[suffix] = buckets_r[i];
	});
	
}

void get_sort_keys(thrust::device_vector<int>& keys, thrust::device_vector<int>& buckets, thrust::device_vector<int>& rank, thrust::device_vector<int>& sa, int step){

	int *rank_r = raw(rank);
	int *sa_r = raw(sa);
	int *keys_r = raw(keys);
	int *buckets_r = raw(buckets);

	auto r = counting_iterator<int>(0);
	int n = keys.size();

	for_each(r, r + n, [=] __device__(int i) {
		//TODO: check if already sorted

		int next_suffix = sa_r[i] + step;
		//Went of end of string - must be lexicographically less than rest of bucket
		if (next_suffix >= n){
			//TODO: can this just be -1?
			keys_r[i] = -next_suffix;
		}
		//Else set sort key to rank of next suffix
		else{
			keys_r[i] = rank_r[next_suffix];
		}
	});


}

void sort_sa(thrust::device_vector<int>& keys, thrust::device_vector<int>& buckets, thrust::device_vector<int>& sa){
	
	stable_sort_by_key(keys.begin(), keys.end(), sa.begin());

	stable_sort_by_key(buckets.begin(), buckets.end(), sa.begin());

}

int suffix_array(const thrust::device_vector<U8>& data, thrust::device_vector<int>& sa){

	assert(data.size() == sa.size());

	int n = data.size();

	//Init suffix array
	sequence(sa.begin(), sa.end());

	device_vector<int> keys(n); //Sort keys
	device_vector<int> buckets(n, 0); //Bucket head flags
	device_vector<int> rank(n); //Rank of suffixes

	copy(data.begin(), data.end(), keys.begin());

	//Radix sort data and SA
	stable_sort_by_key(keys.begin(), keys.end(), sa.begin());

	int step = 1;
	//Begin prefix doubling loop - runs at most log(n) times
	while (true){
		
		//Mark bucket heads
		mark_head(keys, buckets);

		//Check if we are done, i.e. every item is a bucket head
		int result = reduce(buckets.begin(), buckets.end(), INT_MAX, minimum<int>());
		if (result == 1) break;

		//Get rank of suffixes
		get_rank(buckets, rank, sa);
		
		//Use rank as new sort keys
		get_sort_keys(keys, buckets, rank, sa, step);

		//Sort
		sort_sa(keys, buckets, sa);


		print("SA", sa);
		print("Keys", keys);
		print("Buckets", buckets);
		print("rank", rank);
		return 0;

		step *= 2;

		//Just in case, check for infinite loop
		if (step < 0){
			std::cout << "Error: Prefix doubling infinite loop.\n";
			return 1;
		}
	}
	
	return 0;
}


int main()
{
	const size_t n = 6;
	char test[] = "banana";

	device_vector<U8> data(test, test + n); //Input data
	device_vector<int> sa(n); //Suffix array
	
	suffix_array(data, sa);
	
    return 0;
}
