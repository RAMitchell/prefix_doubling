#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>

//Thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/execution_policy.h>

#include "cuda_helpers.cuh"

typedef uint8_t U8;
typedef uint32_t U32;

using namespace thrust;

//Pack 4 U8's into an int
void pack(device_vector<U8>& data, device_vector<int>& keys){
	
	U8 *data_r = raw(data);
	int *keys_r = raw(keys);

	auto r = counting_iterator<int>(0);
	int n = keys.size();

	for_each(r, r + n, [=] __device__(int i) {
		
		int packed = data_r[i];
		
		packed <<= 8;
		if (i + 1 < n)
			packed |= data_r[i+1];
		packed <<= 8;
		if (i + 2 < n)
			packed |= data_r[i + 2];
		packed <<= 8;
		if (i + 3 < n)
			packed |= data_r[i + 3];
		
		keys_r[i] = packed;

	});
}

void mark_head(device_vector<int>& keys, device_vector<U8>& buckets){

	int *keys_r = raw(keys);
	U8 *bucket_r = raw(buckets);
	auto r = counting_iterator<int>(0);
	int n = keys.size();

	for_each(r, r + n, [=] __device__(int i) {
		//Already marked - don't need to do anything
		if (bucket_r[i] == 1){
			return;
		}
		//First item is always head
		else if (i == 0){
			bucket_r[i] = 1;
		}
		//Is different than previous item - must be a bucket head
		else if (keys_r[i] != keys_r[i - 1]){
			bucket_r[i] = 1;
		}
	});
}

void get_rank(device_vector<U8>& buckets, device_vector<int>& b_scan, device_vector<int>& rank, device_vector<int>& sa){

	//Scan bucket heads
	//Copy buckets into rank before we scan it into b_scan - scanning 8 bit types creates problems
	copy(buckets.begin(), buckets.end(), rank.begin());
	inclusive_scan(rank.begin(), rank.end(), b_scan.begin());

	//Calculate rank - stores rank inverse to the suffix array
	// e.g. rank[3] stores the bucket position of sa[?] = 3
	int *rank_r = raw(rank);
	int *sa_r = raw(sa);
	int *b_scan_r = raw(b_scan);

	auto r = counting_iterator<int>(0);
	int n = sa.size();

	for_each(r, r + n, [=] __device__(int i) {
		int suffix = sa_r[i];
		rank_r[suffix] = b_scan_r[i];
	});

}

void get_sort_keys(device_vector<int>& keys, device_vector<int>& rank, device_vector<int>& sa, device_vector<U8>& buckets, int step){

	int *rank_r = raw(rank);
	int *sa_r = raw(sa);
	int *keys_r = raw(keys);
	U8 *buckets_r = raw(buckets);

	auto r = counting_iterator<int>(0);
	int n = keys.size();

	for_each(r, r + n, [=] __device__(int i) {
		//Check if already sorted
		//If is last item - just need to check its flag
		if (buckets_r[i] == 1 && i == n - 1)
			return;
		//Otherwise, if the current item and its next item are flagged, current item must be already sorted
		else if (buckets_r[i] == 1 && buckets_r[i + 1])
			return;

		//Set sort keys
		int next_suffix = sa_r[i] + step;
		//Went off end of string - must be lexicographically less than rest of bucket
		if (next_suffix >= n)
			keys_r[i] = -next_suffix;

		//Else set sort key to rank of next suffix
		else
			keys_r[i] = rank_r[next_suffix];
	});
}


//We have to do a 2 pass sort here to get a "segmented sort"
void sort_sa(device_vector<int>& keys, device_vector<int>& b_scan, device_vector<int>& sa){

	stable_sort_by_key(keys.begin(), keys.end(), make_zip_iterator(make_tuple(sa.begin(), b_scan.begin())));
	stable_sort_by_key(b_scan.begin(), b_scan.end(), make_zip_iterator(make_tuple(sa.begin(), keys.begin())));
}

int device_sa(const unsigned char *data_in, int *sa_in, int n){
	
	try{
		//Copy up to device vectors
		device_vector<U8> data(data_in, data_in + n);
		device_vector<int> sa(n);

		//Init suffix array
		sequence(sa.begin(), sa.end());

		device_vector<int> keys(n); //Sort keys
		device_vector<U8> buckets(n, 0); //Bucket head flags
		device_vector<int> b_scan(n); //Scanned head flags
		device_vector<int> rank(n); //Rank of suffixes

		//Pack 4 bytes into keys so we can radix sort to H order 4 before prefix doubling
		pack(data, keys);
		
		//Radix sort as unsigned 
		//We have to cast keys to a raw pointer then to a device_ptr to convince thrust its unsigned
		unsigned int *keys_r = (unsigned int*)raw(keys);
		device_ptr<unsigned int> keys_ptr(keys_r);
		stable_sort_by_key(keys_ptr, keys_ptr + n, sa.begin());

		int step = 4;
		//Begin prefix doubling loop - runs at most log(n) times
		while (true){

			//Mark bucket heads
			mark_head(keys, buckets);

			//Check if we are done, i.e. every item is a bucket head
			int result = reduce(buckets.begin(), buckets.end(), INT_MAX, minimum<int>());
			if (result == 1) break;

			//Get rank of suffixes
			get_rank(buckets, b_scan, rank, sa);

			//Use rank as new sort keys
			get_sort_keys(keys, rank, sa, buckets, step);

			//Sort
			sort_sa(keys, b_scan, sa);
			/*
			std::cout << "-----\n";
			print("SA", sa);
			print("Keys", keys);
			print("Buckets", buckets);
			print("rank", rank);
			std::cout << "-----\n";
			*/
			step *= 2;

			//Just in case, check for infinite loop
			if (step < 0){
				std::cout << "Error: Prefix doubling infinite loop.\n";
				return 1;
			}
		}

		//Copy SA back to host
		safe_cuda(hipMemcpy(sa_in, raw(sa), sizeof(int)*sa.size(), hipMemcpyDeviceToHost));
	}
	catch (thrust::system_error &e)
	{
		std::cerr << "CUDA error: " << e.what() << std::endl;
	}

	return 0;
}